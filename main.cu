
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdlib>
#include <iostream>

using std::cout;
using std::endl;

#define BLOCKSIZE 50
#define RADIUS 10
#define maxn 2000
#define size maxn*sizeof(int)

__global__ void add(const int *a, const int *b, int *c) {
    c[threadIdx.x + blockIdx.x * blockDim.x] =
            a[threadIdx.x + blockIdx.x * blockDim.x] + b[threadIdx.x + blockIdx.x * blockDim.x];
}

__global__ void stential(const int *in, int *out) {
    __shared__ int tmp[BLOCKSIZE + RADIUS * 2];
    auto gindex = threadIdx.x + blockIdx.x * blockDim.x;
    auto lindex = threadIdx.x + RADIUS;

    tmp[lindex] = in[gindex];
    if (threadIdx.x < RADIUS) {
        tmp[lindex - RADIUS] = (gindex < RADIUS ) ? 0 : in[gindex - RADIUS];
        tmp[lindex + BLOCKSIZE] = (gindex + BLOCKSIZE > maxn) ? 0 : in[gindex + BLOCKSIZE];
    }

    __syncthreads();

    int result = 0;
    for (int i = -RADIUS; i < RADIUS; i++) {
        result += tmp[i + lindex];
    }
    out[gindex] = result;
}

int main() {
    int *in, *out;
    int *inc, *outc;
    in = (int *) malloc(size);
    out = (int *) malloc(size);
    for (int i = 0; i < maxn; i++) {
        in[i] = i;
        out[i] = 233;
    }
    hipMalloc((void **) &inc, size);
    hipMalloc((void **) &outc, size);

    hipMemcpy(inc, in, size, hipMemcpyHostToDevice);
    stential<<<(maxn+BLOCKSIZE-1)/BLOCKSIZE, BLOCKSIZE>>>(inc, outc);
    hipError_t error;
    cout<<size<<" "<<sizeof(out)<<" "<<sizeof(outc)<<endl;
    error = hipMemcpy(out, outc, size, hipMemcpyDeviceToHost);//why i can't copy the memory?
    std::cout << "error: " << hipGetErrorString(error) << std::endl;

    for (int i = 0; i < maxn; i++) {
        std::cout << in[i] << " " << out[i] << std::endl;
    }
    hipFree(inc);
    hipFree(outc);
    return 0;
}